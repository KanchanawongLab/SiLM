#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "GPUgaussLib.cuh"

//gpu refit routine when psf is fit with two sigma sx and sy
//l-m algorithms are used to solve the optimization problem
__global__ void kernel_gpustorm_fitxynbg(float *data,  float *fitresult, float *fiterror,const int iterations, const int sz, float *pold){

	const int NV=2;
	float dudt[NV];
	//printf("%d\n",sz);

	int tx=threadIdx.x;
	int bx=blockIdx.x;
	int BlockSize=blockDim.x;

	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	
	//pold  x y N bg sigma


	float NR_Numerator[NV], NR_Denominator[NV];
	float theta[NV];
	float maxjump[NV]={100.0f, 2.0f};
	float gamma[NV]={ 0.5f, 1.0f};
	

	const float *sdata=data+sz*sz*(bx*BlockSize+tx);
    float *spold=pold+6*(bx*BlockSize+tx);
	const int sz1=sz;
	//initial value of fitting I, bg,
	float psfSigma=spold[4];
	float Nmax=0;
	kernel_GaussFMaxMin2D(sz, psfSigma, sdata, &Nmax, &theta[1]);
	theta[0]=kernel_cumax(0.0, (Nmax-theta[1])*2*pi*psfSigma*psfSigma);
	theta[1] = kernel_cumax(theta[1],0.01);
	
	
	maxjump[0]=kernel_cumax(maxjump[0],theta[0]);
    maxjump[1]=kernel_cumax(maxjump[1],theta[1]);


	float new_err=1e13; 
	float old_err=1e15;
	float lambda=0.1, scale_up=10,scale_down=0.1,acceptance=1.1;
	float new_lambda=lambda;
	float old_lambda;
	float tolerance=1e-6;
	float new_update[NV]={1e13f,1e13f};
	float old_update[NV];
	float old_theta[NV];

	float M_jacob[NV]={0};
	float M_hession[NV][NV]={1};
	float Minv_hession[NV][NV];
	float diagMinv_hession[NV];
	int p=0; int q=0;
	
	


	float model, modela,datai;
	float zfact1;
	int m=0; float t1=0.0f,t2=0.0f; int mm=0;
	

	float mu=1+new_lambda;
	int kk1=0;

	

	for (kk1=0;kk1<iterations;kk1++)
	{
		
	  memset(M_jacob,0,NV*sizeof(float));
	  memset(M_hession,0,NV*NV*sizeof(float));
		for (int jj=0;jj<NV; jj++)
		{
			old_update[jj]=new_update[jj];
			old_theta[jj]=theta[jj];
		}
		old_lambda=new_lambda;
		old_err=new_err;
		
		for (m=0; m<sz;m++) for (mm=0;mm<sz;mm++) {

			kernel_DerivativeGauss2D_nbg(m,mm,theta,spold,dudt,&model);

			datai=sdata[sz*mm+m];
			
			
			if (model>10e-3){
				t1=1-datai/model;
				t2=datai/pow(model,2);}

			for (p=0; p<NV;p++)
			{
				M_jacob[p]=M_jacob[p]+t1*dudt[p];
				for(q=0; q<NV;q++)
				{
					M_hession[p][q]=M_hession[p][q]+t2*dudt[p]*dudt[q];
				}
			}

			}
		
		for (p=0;p<NV;p++){
			M_hession[p][p]=M_hession[p][p]+(mu-1);}

		kernel_MatInvN(*M_hession, *Minv_hession, diagMinv_hession, NV) ;

       for (p=0; p<NV;p++)
		{
				new_update[p]=0;
				for(q=0; q<NV;q++)
				{
					new_update[p]=new_update[p]+Minv_hession[p][q]*M_jacob[q];

				}
		}
		
		for (p=0;p<NV;p++){
			//new_update[p]=M_jacob[p]/diagMinv_hession[p];
			if (new_update[p]/old_update[p]<-0.5)
				maxjump[p]=0.5*maxjump[p];

			new_update[p]=new_update[p]/(1+abs(new_update[p]/maxjump[p]));
			theta[p]=theta[p]-new_update[p];
		}

	   
		
		theta[0] = kernel_cumax(theta[0],1.0);
		theta[1] = kernel_cumax(theta[1],0.01);
		

		

	
		new_err=0;
		for(m=0; m<sz;m++) for(mm=0;mm<sz;mm++){
			
			kernel_DerivativeGauss2D_nbg(m,mm,theta,spold, dudt,&model);
			datai=sdata[sz*mm+m];
			if (datai>0)
				new_err=new_err+2*((model-datai)-datai*log(model/datai));
			else
				new_err=new_err+2*model;
		}

	

		if (new_err>acceptance*old_err){
			for (p=0;p<NV;p++){
				theta[p]=old_theta[p];
				new_update[p]=old_update[p];
			}
			new_lambda=old_lambda;
			new_err=old_err;
			mu=kernel_cumax((1+new_lambda*scale_up)/(1+new_lambda),1.3);
			new_lambda=scale_up*new_lambda;}

		if (new_err<old_err){
			new_lambda=scale_down*new_lambda;
			mu=1+new_lambda;}

		fiterror[(tid)*iterations+kk1]=new_err;
		
	}

	
    //calculating the crlb and new_err
	float Mcrlb[NV][NV]={0}; float Mcrlb_inv[NV][NV];
	float diagMcrlb_inv[NV];
	new_err=0;
	for (m=0; m<sz;m++) for(mm=0;mm<sz;mm++){
            
			kernel_DerivativeGauss2D_nbg(m,mm,theta,spold, dudt,&model);
			datai=sdata[sz*mm+m];

			if (datai>0)
				new_err=new_err+2*((model-datai)-datai*log(model/datai));
			else
				new_err=new_err+2*model;

			for (p=0;p<NV;p++)
				for(q=0;q<NV;q++)
				{
					{
						Mcrlb[p][q]=Mcrlb[p][q]+dudt[p]*dudt[q]/(model);
					}
				}

		   
	}

	kernel_MatInvN(*Mcrlb, *Mcrlb_inv, diagMcrlb_inv, NV) ; 


	//output fit result
	for (m = 0; m < NV; m++) {
		fitresult[(tid)*NV + m] = theta[m];
	}
	


	//return;  
}
       
